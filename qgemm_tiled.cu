#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <vector>
#include <stdint.h>
#include <assert.h>

// ------------------- Constants -------------------
#define TILE_SIZE 32
#define REG_TILE_N 4
#define REG_TILE_M 4

// ------------------- User kernels (mostly as provided) -------------------

__global__ void absmax_rowwise_kernel(const float* __restrict__ A, float* rowScales, int M, int K) {
    int m = blockIdx.x;
    if (m < M) {
        extern __shared__ float sdata[];
        int tid = threadIdx.x;
        float local_max = 0.f;
        for (int k = tid; k < K; k += blockDim.x) {
            float val = fabsf(A[m * K + k]);
            local_max = fmaxf(local_max, val);
        }
        sdata[tid] = local_max;
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
            }
            __syncthreads();
        }
        if (tid == 0) {
            rowScales[m] = fmaxf(sdata[0] / 127.f, 1e-8f);
        }
    }
}

__global__ void colwise_minmax(const float* __restrict__ A, float* colMins, float* colMaxs, int M, int N) {
    int n = blockIdx.x;
    if (n < N) {
        extern __shared__ float sdata[];
        float* smins = sdata;
        float* smaxs = sdata + blockDim.x;

        int tid = threadIdx.x;
        float local_min = 1e30f;
        float local_max = -1e30f;
        for (int i = tid; i < M; i += blockDim.x) {
            float val = A[i * N + n];
            local_min = fminf(local_min, val);
            local_max = fmaxf(local_max, val);
        }
        smins[tid] = local_min;
        smaxs[tid] = local_max;
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                smins[tid] = fminf(smins[tid], smins[tid + s]);
                smaxs[tid] = fmaxf(smaxs[tid], smaxs[tid + s]);
            }
            __syncthreads();
        }
        if (tid == 0) {
            colMins[n] = smins[0];
            colMaxs[n] = smaxs[0];
        }
    }
}

__global__ void quantize_weights_rowwise(const float* __restrict__ A, int8_t* Aq,
                                        const float* __restrict__ rowScales, int M, int K) {
    int m = blockIdx.x;
    int k = threadIdx.x;
    if (m < M && k < K) {
        float scale = rowScales[m];
        float val = A[m * K + k] / scale;
        val = fmaxf(fminf(val, 127.f), -127.f);
        Aq[m * K + k] = static_cast<int8_t>(rintf(val));
    }
}

__global__ void quantize_activations_colwise(const float* __restrict__ X, uint8_t* Xq,
                                            const float* __restrict__ colMins,
                                            const float* __restrict__ colMaxs,
                                            float* colScales, int* colZPs,
                                            int M, int N) {
    int n = blockIdx.x;
    int m = threadIdx.x;
    if (n < N && m < M) {
        float minv = colMins[n];
        float maxv = colMaxs[n];
        float scale = fmaxf((maxv - minv) / 255.f, 1e-8f);
        int zp = static_cast<int>(rintf(-minv / scale));
        colScales[n] = scale;
        colZPs[n] = zp;

        float val = X[m * N + n] / scale + zp;
        val = fmaxf(fminf(val, 255.f), 0.f);
        Xq[m * N + n] = static_cast<uint8_t>(rintf(val));
    }
}

__global__ void qgemm_kernel(const int8_t* __restrict__ Wq,
                                                 const float* __restrict__ Sw,
                                                 const uint8_t* __restrict__ Xq,
                                                 const float* __restrict__ Sx,
                                                 const int* __restrict__ Zx,
                                                 float* Y, int M, int K, int N) {
    extern __shared__ int8_t smem_raw[]; // optional if dynamic shared needed
    // static shared arrays (compile-time sizes) - preferred
    __shared__ int8_t  Wq_tile[TILE_SIZE][TILE_SIZE];
    __shared__ int32_t Xq_tile[TILE_SIZE][TILE_SIZE];

    const int bx = blockIdx.x, by = blockIdx.y;
    const int tx = threadIdx.x, ty = threadIdx.y;

    const int base_row = by * TILE_SIZE;
    const int base_col = bx * TILE_SIZE;

    // thread-owned offset within the tile
    const int thread_row_in_tile = ty * REG_TILE_M; // 0..TILE_SIZE-REG_TILE_M
    const int thread_col_in_tile = tx * REG_TILE_N; // 0..TILE_SIZE-REG_TILE_N

    const int global_row = base_row + thread_row_in_tile;
    const int global_col = base_col + thread_col_in_tile;

    int32_t acc[REG_TILE_M][REG_TILE_N];
    #pragma unroll
    for (int i = 0; i < REG_TILE_M; ++i)
        #pragma unroll
        for (int j = 0; j < REG_TILE_N; ++j)
            acc[i][j] = 0;

    int8_t w_reg[REG_TILE_M];

    const int numTiles = (K + TILE_SIZE - 1) / TILE_SIZE;

    for (int t = 0; t < numTiles; ++t) {
        // how many K values are valid in this tile
        const int K_tile_valid = min(TILE_SIZE, K - t * TILE_SIZE);

        // Cooperative load: each thread writes REG_TILE_M x REG_TILE_N entries
        #pragma unroll
        for (int lm = 0; lm < REG_TILE_M; ++lm) {
            #pragma unroll
            for (int ln = 0; ln < REG_TILE_N; ++ln) {
                int tile_r = thread_row_in_tile + lm;   // 0..TILE_SIZE-1
                int tile_c = thread_col_in_tile + ln;   // 0..TILE_SIZE-1

                int Wq_row = base_row + tile_r;         // global m
                int Wq_col = t * TILE_SIZE + tile_c;    // global k

                if (Wq_row < M && Wq_col < K) {
                    Wq_tile[tile_r][tile_c] = Wq[Wq_row * K + Wq_col];
                } else {
                    // MUST zero any shared cell that could be read later
                    Wq_tile[tile_r][tile_c] = 0;
                }

                int Xq_row = t * TILE_SIZE + tile_r;    // global k
                int Xq_col = base_col + tile_c;         // global n
                if (Xq_row < K && Xq_col < N) {
                    int32_t xval = static_cast<int32_t>(Xq[Xq_row * N + Xq_col]);
                    Xq_tile[tile_r][tile_c] = xval - Zx[Xq_col];
                } else {
                    Xq_tile[tile_r][tile_c] = 0;
                }
            }
        }

        __syncthreads();

        // Compute only up to K_tile_valid k-steps; but we still iterate TILE_SIZE,
        // reading shared entries — they were zeroed above for invalid K positions.
        for (int k = 0; k < TILE_SIZE; ++k) {
            // If k >= K_tile_valid then Xq_tile[k][*] and Wq_tile[*][k] are zeros by construction.
            #pragma unroll
            for (int i = 0; i < REG_TILE_M; ++i) {
                w_reg[i] = Wq_tile[thread_row_in_tile + i][k];
            }

            #pragma unroll
            for (int i = 0; i < REG_TILE_M; ++i) {
                #pragma unroll
                for (int j = 0; j < REG_TILE_N; ++j) {
                    acc[i][j] += static_cast<int32_t>(w_reg[i]) * Xq_tile[k][thread_col_in_tile + j];
                }
            }
        }

        __syncthreads();
    }

    // Write back with scaling
    #pragma unroll
    for (int i = 0; i < REG_TILE_M; ++i) {
        #pragma unroll
        for (int j = 0; j < REG_TILE_N; ++j) {
            int out_r = global_row + i;
            int out_c = global_col + j;
            if (out_r < M && out_c < N) {
                Y[out_r * N + out_c] = static_cast<float>(acc[i][j]) * Sw[out_r] * Sx[out_c];
            }
        }
    }
}

// ------------------- C++ / PyTorch wrappers -------------------

inline void check_contiguous_cuda(const torch::Tensor &t, const char* name) {
    TORCH_CHECK(t.device().is_cuda(), "%s must be a CUDA tensor", name);
    TORCH_CHECK(t.is_contiguous(), "%s must be contiguous", name);
}

torch::Tensor absmax_rowwise(torch::Tensor A) {
    check_contiguous_cuda(A, "A");
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    int64_t M = A.size(0);
    int64_t K = A.size(1);

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(A.device());
    torch::Tensor rowScales = torch::empty({M}, options);

    // launch kernel: grid.x = M, block.x = min(1024, K) but >= 1
    int threads = static_cast<int>(std::min<int64_t>(1024, K > 0 ? K : 1));
    if (threads < 1) threads = 1;
    dim3 blocks(static_cast<uint32_t>(M));
    size_t shared_bytes = threads * sizeof(float);

    float* A_ptr = A.data_ptr<float>();
    float* out_ptr = rowScales.data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    absmax_rowwise_kernel<<<blocks, threads, shared_bytes, stream>>>(A_ptr, out_ptr, (int)M, (int)K);
    AT_CUDA_CHECK(hipGetLastError());
    return rowScales;
}

std::vector<torch::Tensor> colwise_minmax_wrapper(torch::Tensor A) {
    check_contiguous_cuda(A, "A");
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    int64_t M = A.size(0);
    int64_t N = A.size(1);

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(A.device());
    torch::Tensor colMins = torch::empty({N}, options);
    torch::Tensor colMaxs = torch::empty({N}, options);

    int threads = static_cast<int>(std::min<int64_t>(1024, M > 0 ? M : 1));
    if (threads < 1) threads = 1;
    dim3 blocks(static_cast<uint32_t>(N));
    size_t shared_bytes = threads * 2 * sizeof(float);

    float* A_ptr = A.data_ptr<float>();
    float* mins_ptr = colMins.data_ptr<float>();
    float* maxs_ptr = colMaxs.data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    colwise_minmax<<<blocks, threads, shared_bytes, stream>>>(A_ptr, mins_ptr, maxs_ptr, (int)M, (int)N);
    AT_CUDA_CHECK(hipGetLastError());
    return {colMins, colMaxs};
}

torch::Tensor quantize_weights_rowwise_wrapper(torch::Tensor A, torch::Tensor rowScales) {
    check_contiguous_cuda(A, "A");
    check_contiguous_cuda(rowScales, "rowScales");
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    TORCH_CHECK(rowScales.dim() == 1, "rowScales must be 1D");
    int64_t M = A.size(0);
    int64_t K = A.size(1);
    TORCH_CHECK(rowScales.size(0) == M, "rowScales length must equal M");

    auto options = torch::TensorOptions().dtype(torch::kInt8).device(A.device());
    torch::Tensor Aq = torch::empty({M, K}, options);

    int threads = static_cast<int>(std::min<int64_t>(1024, K > 0 ? K : 1));
    if (threads < 1) threads = 1;
    dim3 blocks(static_cast<uint32_t>(M));

    float* A_ptr = A.data_ptr<float>();
    int8_t* Aq_ptr = reinterpret_cast<int8_t*>(Aq.data_ptr<int8_t>());
    float* scales_ptr = rowScales.data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    quantize_weights_rowwise<<<blocks, threads, 0, stream>>>(A_ptr, Aq_ptr, scales_ptr, (int)M, (int)K);
    AT_CUDA_CHECK(hipGetLastError());
    return Aq;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> quantize_activations_colwise_wrapper(torch::Tensor X) {
    check_contiguous_cuda(X, "X");
    TORCH_CHECK(X.dim() == 2, "X must be 2D");
    int64_t M = X.size(0);
    int64_t N = X.size(1);

    auto options_u8 = torch::TensorOptions().dtype(torch::kUInt8).device(X.device());
    auto options_f = torch::TensorOptions().dtype(torch::kFloat32).device(X.device());
    auto options_i = torch::TensorOptions().dtype(torch::kInt32).device(X.device());

    torch::Tensor Xq = torch::empty({M, N}, options_u8);
    torch::Tensor colScales = torch::empty({N}, options_f);
    torch::Tensor colZPs = torch::empty({N}, options_i);

    int threads = static_cast<int>(std::min<int64_t>(1024, M > 0 ? M : 1));
    if (threads < 1) threads = 1;
    dim3 blocks(static_cast<uint32_t>(N));

    float* X_ptr = X.data_ptr<float>();
    uint8_t* Xq_ptr = Xq.data_ptr<uint8_t>();
    float* mins_ptr; // we need mins/maxs to compute scales -> call colwise_minmax first
    float* maxs_ptr;

    // compute col mins/maxs
    torch::Tensor colMins = torch::empty({N}, options_f);
    torch::Tensor colMaxs = torch::empty({N}, options_f);
    {
        int threads_mm = static_cast<int>(std::min<int64_t>(1024, M > 0 ? M : 1));
        size_t shared_bytes = threads_mm * 2 * sizeof(float);
        hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
        colwise_minmax<<<dim3((uint32_t)N), threads_mm, shared_bytes, stream>>>(X_ptr, colMins.data_ptr<float>(), colMaxs.data_ptr<float>(), (int)M, (int)N);
        AT_CUDA_CHECK(hipGetLastError());
    }

    mins_ptr = colMins.data_ptr<float>();
    maxs_ptr = colMaxs.data_ptr<float>();
    float* scales_ptr = colScales.data_ptr<float>();
    int* zps_ptr = colZPs.data_ptr<int>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    quantize_activations_colwise<<<dim3((uint32_t)N), threads, 0, stream>>>(X_ptr, Xq_ptr, mins_ptr, maxs_ptr, scales_ptr, zps_ptr, (int)M, (int)N);
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(Xq, colScales, colZPs);
}

torch::Tensor qgemm_wrapper(torch::Tensor Wq, torch::Tensor Sw, torch::Tensor Xq, torch::Tensor Sx, torch::Tensor Zx, int M, int K, int N) {
    // Validate
    check_contiguous_cuda(Wq, "Wq");
    check_contiguous_cuda(Sw, "Sw");
    check_contiguous_cuda(Xq, "Xq");
    check_contiguous_cuda(Sx, "Sx");
    check_contiguous_cuda(Zx, "Zx");

    TORCH_CHECK(Wq.dim() == 2 && Wq.size(0) == M && Wq.size(1) == K, "Wq must be MxK int8");
    TORCH_CHECK(Sw.dim() == 1 && Sw.size(0) == M, "Sw must be length M");
    TORCH_CHECK(Xq.dim() == 2 && Xq.size(0) == K && Xq.size(1) == N, "Xq must be KxN uint8");
    TORCH_CHECK(Sx.dim() == 1 && Sx.size(0) == N, "Sx must be length N");
    TORCH_CHECK(Zx.dim() == 1 && Zx.size(0) == N, "Zx must be length N");

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(Wq.device());
    torch::Tensor Y = torch::zeros({M, N}, options);

    // Use register-tiled kernel configuration
    // Each thread block handles TILE_SIZE x TILE_SIZE elements
    // Each thread handles REG_TILE_M x REG_TILE_N elements
    const int threads_per_block_x = TILE_SIZE / REG_TILE_N;  // 32/4 = 8
    const int threads_per_block_y = TILE_SIZE / REG_TILE_M;  // 32/4 = 8
    
    dim3 block(threads_per_block_x, threads_per_block_y);  // 8x8 = 64 threads per block
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    // Calculate shared memory size needed for the tiles
    size_t shared_mem_size = TILE_SIZE * TILE_SIZE * sizeof(int8_t) +  // Wq_tile
                            TILE_SIZE * TILE_SIZE * sizeof(int32_t);   // Xq_tile

    // pointers
    const int8_t* Wq_ptr = reinterpret_cast<const int8_t*>(Wq.data_ptr<int8_t>());
    const float* Sw_ptr = Sw.data_ptr<float>();
    const uint8_t* Xq_ptr = Xq.data_ptr<uint8_t>();
    const float* Sx_ptr = Sx.data_ptr<float>();
    const int* Zx_ptr = Zx.data_ptr<int>();
    float* Y_ptr = Y.data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    qgemm_kernel<<<grid, block, shared_mem_size, stream>>>(Wq_ptr, Sw_ptr, Xq_ptr, Sx_ptr, Zx_ptr, Y_ptr, M, K, N);
    AT_CUDA_CHECK(hipGetLastError());
    return Y;
}

// ------------------- Python binding -------------------

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.doc() = "Quantization ops (CUDA) - wrappers for custom kernels";

    m.def("absmax_rowwise", &absmax_rowwise, "Compute row-wise absolute max and produce rowScales (float32) (CUDA)");
    m.def("colwise_minmax", &colwise_minmax_wrapper, "Compute column min/max (returns (mins,maxs)) (CUDA)");
    m.def("quantize_weights_rowwise", &quantize_weights_rowwise_wrapper, "Quantize weights rowwise to int8 (CUDA)");
    m.def("quantize_activations_colwise", &quantize_activations_colwise_wrapper, "Quantize activations columnwise -> returns (Xq, colScales, colZPs) (CUDA)");
    m.def("qgemm", &qgemm_wrapper, "Quantized GEMM with register tiling (CUDA). Call as qgemm(Wq, Sw, Xq, Sx, Zx, M, K, N)");
}