#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <vector>
#include <stdint.h>
#include <assert.h>

// ------------------- Constants -------------------
#define TILE_SIZE 32
#define REG_TILE_N 4
#define REG_TILE_M 4

// ------------------- User kernels (mostly as provided) -------------------

__global__ void absmax_rowwise_kernel(const float* __restrict__ A, float* rowScales, int M, int K) {
    int m = blockIdx.x;
    if (m < M) {
        extern __shared__ float sdata[];
        int tid = threadIdx.x;
        float local_max = 0.f;
        for (int k = tid; k < K; k += blockDim.x) {
            float val = fabsf(A[m * K + k]);
            local_max = fmaxf(local_max, val);
        }
        sdata[tid] = local_max;
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
            }
            __syncthreads();
        }
        if (tid == 0) {
            rowScales[m] = fmaxf(sdata[0] / 127.f, 1e-8f);
        }
    }
}

__global__ void colwise_minmax(const float* __restrict__ A, float* colMins, float* colMaxs, int M, int N) {
    int n = blockIdx.x;
    if (n < N) {
        extern __shared__ float sdata[];
        float* smins = sdata;
        float* smaxs = sdata + blockDim.x;

        int tid = threadIdx.x;
        float local_min = 1e30f;
        float local_max = -1e30f;
        for (int i = tid; i < M; i += blockDim.x) {
            float val = A[i * N + n];
            local_min = fminf(local_min, val);
            local_max = fmaxf(local_max, val);
        }
        smins[tid] = local_min;
        smaxs[tid] = local_max;
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                smins[tid] = fminf(smins[tid], smins[tid + s]);
                smaxs[tid] = fmaxf(smaxs[tid], smaxs[tid + s]);
            }
            __syncthreads();
        }
        if (tid == 0) {
            colMins[n] = smins[0];
            colMaxs[n] = smaxs[0];
        }
    }
}

__global__ void quantize_weights_rowwise(const float* __restrict__ A, int8_t* Aq,
                                        const float* __restrict__ rowScales, int M, int K) {
    int m = blockIdx.x;
    int k = threadIdx.x;
    if (m < M && k < K) {
        float scale = rowScales[m];
        float val = A[m * K + k] / scale;
        val = fmaxf(fminf(val, 127.f), -127.f);
        Aq[m * K + k] = static_cast<int8_t>(rintf(val));
    }
}

__global__ void quantize_activations_colwise(const float* __restrict__ X, uint8_t* Xq,
                                            const float* __restrict__ colMins,
                                            const float* __restrict__ colMaxs,
                                            float* colScales, int* colZPs,
                                            int M, int N) {
    int n = blockIdx.x;
    int m = threadIdx.x;
    if (n < N && m < M) {
        float minv = colMins[n];
        float maxv = colMaxs[n];
        float scale = fmaxf((maxv - minv) / 255.f, 1e-8f);
        int zp = static_cast<int>(rintf(-minv / scale));
        colScales[n] = scale;
        colZPs[n] = zp;

        float val = X[m * N + n] / scale + zp;
        val = fmaxf(fminf(val, 255.f), 0.f);
        Xq[m * N + n] = static_cast<uint8_t>(rintf(val));
    }
}

__global__ void qgemm_kernel(const int8_t* __restrict__ Wq, const float* __restrict__ Sw,
                            const uint8_t* __restrict__ Xq, const float* __restrict__ Sx,
                            const int* __restrict__ Zx, float* Y, int M, int K, int N) {
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (m < M && n < N) {
        int32_t acc = 0;
        for (int k = 0; k < K; k++) {
            int32_t w = static_cast<int32_t>(Wq[m * K + k]);
            int32_t x = static_cast<int32_t>(Xq[k * N + n]) - Zx[n];
            acc += w * x;
        }
        float result = acc * (Sw[m] * Sx[n]);
        Y[m * N + n] = result;
    }
}

// ------------------- C++ / PyTorch wrappers -------------------

inline void check_contiguous_cuda(const torch::Tensor &t, const char* name) {
    TORCH_CHECK(t.device().is_cuda(), "%s must be a CUDA tensor", name);
    TORCH_CHECK(t.is_contiguous(), "%s must be contiguous", name);
}

torch::Tensor absmax_rowwise(torch::Tensor A) {
    check_contiguous_cuda(A, "A");
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    int64_t M = A.size(0);
    int64_t K = A.size(1);

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(A.device());
    torch::Tensor rowScales = torch::empty({M}, options);

    // launch kernel: grid.x = M, block.x = min(1024, K) but >= 1
    int threads = static_cast<int>(std::min<int64_t>(1024, K > 0 ? K : 1));
    if (threads < 1) threads = 1;
    dim3 blocks(static_cast<uint32_t>(M));
    size_t shared_bytes = threads * sizeof(float);

    float* A_ptr = A.data_ptr<float>();
    float* out_ptr = rowScales.data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    absmax_rowwise_kernel<<<blocks, threads, shared_bytes, stream>>>(A_ptr, out_ptr, (int)M, (int)K);
    AT_CUDA_CHECK(hipGetLastError());
    return rowScales;
}

std::vector<torch::Tensor> colwise_minmax_wrapper(torch::Tensor A) {
    check_contiguous_cuda(A, "A");
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    int64_t M = A.size(0);
    int64_t N = A.size(1);

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(A.device());
    torch::Tensor colMins = torch::empty({N}, options);
    torch::Tensor colMaxs = torch::empty({N}, options);

    int threads = static_cast<int>(std::min<int64_t>(1024, M > 0 ? M : 1));
    if (threads < 1) threads = 1;
    dim3 blocks(static_cast<uint32_t>(N));
    size_t shared_bytes = threads * 2 * sizeof(float);

    float* A_ptr = A.data_ptr<float>();
    float* mins_ptr = colMins.data_ptr<float>();
    float* maxs_ptr = colMaxs.data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    colwise_minmax<<<blocks, threads, shared_bytes, stream>>>(A_ptr, mins_ptr, maxs_ptr, (int)M, (int)N);
    AT_CUDA_CHECK(hipGetLastError());
    return {colMins, colMaxs};
}

torch::Tensor quantize_weights_rowwise_wrapper(torch::Tensor A, torch::Tensor rowScales) {
    check_contiguous_cuda(A, "A");
    check_contiguous_cuda(rowScales, "rowScales");
    TORCH_CHECK(A.dim() == 2, "A must be 2D");
    TORCH_CHECK(rowScales.dim() == 1, "rowScales must be 1D");
    int64_t M = A.size(0);
    int64_t K = A.size(1);
    TORCH_CHECK(rowScales.size(0) == M, "rowScales length must equal M");

    auto options = torch::TensorOptions().dtype(torch::kInt8).device(A.device());
    torch::Tensor Aq = torch::empty({M, K}, options);

    int threads = static_cast<int>(std::min<int64_t>(1024, K > 0 ? K : 1));
    if (threads < 1) threads = 1;
    dim3 blocks(static_cast<uint32_t>(M));

    float* A_ptr = A.data_ptr<float>();
    int8_t* Aq_ptr = reinterpret_cast<int8_t*>(Aq.data_ptr<int8_t>());
    float* scales_ptr = rowScales.data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    quantize_weights_rowwise<<<blocks, threads, 0, stream>>>(A_ptr, Aq_ptr, scales_ptr, (int)M, (int)K);
    AT_CUDA_CHECK(hipGetLastError());
    return Aq;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> quantize_activations_colwise_wrapper(torch::Tensor X) {
    check_contiguous_cuda(X, "X");
    TORCH_CHECK(X.dim() == 2, "X must be 2D");
    int64_t M = X.size(0);
    int64_t N = X.size(1);

    auto options_u8 = torch::TensorOptions().dtype(torch::kUInt8).device(X.device());
    auto options_f = torch::TensorOptions().dtype(torch::kFloat32).device(X.device());
    auto options_i = torch::TensorOptions().dtype(torch::kInt32).device(X.device());

    torch::Tensor Xq = torch::empty({M, N}, options_u8);
    torch::Tensor colScales = torch::empty({N}, options_f);
    torch::Tensor colZPs = torch::empty({N}, options_i);

    int threads = static_cast<int>(std::min<int64_t>(1024, M > 0 ? M : 1));
    if (threads < 1) threads = 1;
    dim3 blocks(static_cast<uint32_t>(N));

    float* X_ptr = X.data_ptr<float>();
    uint8_t* Xq_ptr = Xq.data_ptr<uint8_t>();
    float* mins_ptr; // we need mins/maxs to compute scales -> call colwise_minmax first
    float* maxs_ptr;

    // compute col mins/maxs
    torch::Tensor colMins = torch::empty({N}, options_f);
    torch::Tensor colMaxs = torch::empty({N}, options_f);
    {
        int threads_mm = static_cast<int>(std::min<int64_t>(1024, M > 0 ? M : 1));
        size_t shared_bytes = threads_mm * 2 * sizeof(float);
        hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
        colwise_minmax<<<dim3((uint32_t)N), threads_mm, shared_bytes, stream>>>(X_ptr, colMins.data_ptr<float>(), colMaxs.data_ptr<float>(), (int)M, (int)N);
        AT_CUDA_CHECK(hipGetLastError());
    }

    mins_ptr = colMins.data_ptr<float>();
    maxs_ptr = colMaxs.data_ptr<float>();
    float* scales_ptr = colScales.data_ptr<float>();
    int* zps_ptr = colZPs.data_ptr<int>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    quantize_activations_colwise<<<dim3((uint32_t)N), threads, 0, stream>>>(X_ptr, Xq_ptr, mins_ptr, maxs_ptr, scales_ptr, zps_ptr, (int)M, (int)N);
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(Xq, colScales, colZPs);
}

torch::Tensor qgemm_wrapper(torch::Tensor Wq, torch::Tensor Sw, torch::Tensor Xq, torch::Tensor Sx, torch::Tensor Zx, int M, int K, int N) {
    // Validate
    check_contiguous_cuda(Wq, "Wq");
    check_contiguous_cuda(Sw, "Sw");
    check_contiguous_cuda(Xq, "Xq");
    check_contiguous_cuda(Sx, "Sx");
    check_contiguous_cuda(Zx, "Zx");

    TORCH_CHECK(Wq.dim() == 2 && Wq.size(0) == M && Wq.size(1) == K, "Wq must be MxK int8");
    TORCH_CHECK(Sw.dim() == 1 && Sw.size(0) == M, "Sw must be length M");
    TORCH_CHECK(Xq.dim() == 2 && Xq.size(0) == K && Xq.size(1) == N, "Xq must be KxN uint8");
    TORCH_CHECK(Sx.dim() == 1 && Sx.size(0) == N, "Sx must be length N");
    TORCH_CHECK(Zx.dim() == 1 && Zx.size(0) == N, "Zx must be length N");

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(Wq.device());
    torch::Tensor Y = torch::zeros({M, N}, options);

    // Use simple 2D grid launch instead of register tiling for now
    dim3 block(16, 16);  // 256 threads per block
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    // pointers
    const int8_t* Wq_ptr = reinterpret_cast<const int8_t*>(Wq.data_ptr<int8_t>());
    const float* Sw_ptr = Sw.data_ptr<float>();
    const uint8_t* Xq_ptr = Xq.data_ptr<uint8_t>();
    const float* Sx_ptr = Sx.data_ptr<float>();
    const int* Zx_ptr = Zx.data_ptr<int>();
    float* Y_ptr = Y.data_ptr<float>();

    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    qgemm_kernel<<<grid, block, 0, stream>>>(Wq_ptr, Sw_ptr, Xq_ptr, Sx_ptr, Zx_ptr, Y_ptr, M, K, N);
    AT_CUDA_CHECK(hipGetLastError());
    return Y;
}

// ------------------- Python binding -------------------

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.doc() = "Quantization ops (CUDA) - wrappers for custom kernels";

    m.def("absmax_rowwise", &absmax_rowwise, "Compute row-wise absolute max and produce rowScales (float32) (CUDA)");
    m.def("colwise_minmax", &colwise_minmax_wrapper, "Compute column min/max (returns (mins,maxs)) (CUDA)");
    m.def("quantize_weights_rowwise", &quantize_weights_rowwise_wrapper, "Quantize weights rowwise to int8 (CUDA)");
    m.def("quantize_activations_colwise", &quantize_activations_colwise_wrapper, "Quantize activations columnwise -> returns (Xq, colScales, colZPs) (CUDA)");
    m.def("qgemm", &qgemm_wrapper, "Quantized GEMM with register tiling (CUDA). Call as qgemm(Wq, Sw, Xq, Sx, Zx, M, K, N)");
}
